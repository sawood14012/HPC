
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_SIZE 3
#define WA 3
#define HA 3
#define WB 3
#define HB 3
#define WC 3
#define HC 3

void Init(float * data ,int size)
{
	for(int i = 0; i < size; ++i)
    	data[i] = i;
}

__global__ void matrixMul(float* A,float* B,float* C,int wA,int wB)
{
int tx = threadIdx.x;
int ty = threadIdx.y;

float value = 0;
for(int i = 0; i < wA; ++i)
{
float elementA = A[ty * wA + i];
float elementB = B[i * wB + tx];
value  +=  elementA * elementB;
}
//  write to device mem
C[ty * wA + tx] = value;
}

int main(int argc ,char** argv)
{

srand(2006);

unsigned  int size_A = WA * HA;
unsigned  int mem_size_A =sizeof(float) * size_A;
float* h_A = (float*)  malloc(mem_size_A);

unsigned  int size_B = WB * HB;
unsigned  int mem_size_B =sizeof(float) * size_B;
float * h_B = (float*)  malloc(mem_size_B);

unsigned  int size_C = WC * HC;
unsigned  int mem_size_C =sizeof(float) * size_C;
float * h_C = (float *)  malloc(mem_size_C);

Init(h_A, size_A);
Init(h_B, size_B);

printf("\n\nMatrix A\n");
for(int i = 0; i < size_A; i++)
{
printf("%f ", h_A[i]);
if(((i + 1) % WA) == 0)
printf("\n");
}

printf("\n\nMatrix B\n");
for(int i = 0; i < size_B; i++)
{
printf
("%f ", h_B[i]);
if(((i + 1) % WB) == 0)
printf("\n");
}

float* d_A;
float* d_B;
float* d_C;

hipMalloc((void**) &d_A, mem_size_A);
hipMalloc((void**) &d_B, mem_size_B);
hipMalloc((void**) &d_C, mem_size_C);

hipMemcpy(d_A, h_A,mem_size_A ,hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B,mem_size_B ,hipMemcpyHostToDevice);


dim3  threads(BLOCK_SIZE , BLOCK_SIZE);
dim3  grid(WC / threads.x, HC / threads.y);

matrixMul<<< grid , threads  >>>(d_A,d_B, d_C, WA, WB);
hipMemcpy(h_C, d_C, mem_size_C ,hipMemcpyDeviceToHost);

printf("\n\nMatrix C (Results) \n");
for(int i = 0;i<size_C; i ++){
	printf("%f ",h_C[i]);
	if(((i+ 1) % WC) == 0)
    	printf("\n");
}
printf("\n");

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

free(h_A);
free(h_B);
free(h_C);

}
