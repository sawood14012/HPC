
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void LM(float in)
{
//	float f;    
//	f = in; 	 
}

__global__ void GM(float *array)
{
	array[threadIdx.x] = 2.0f * (float) threadIdx.x;
}

__global__ void SM(float *array)
{
	int i, index = threadIdx.x;
	float average, sum = 0.0f;
     	__shared__ float sh_arr[128];
	sh_arr[index] = array[index];
	__syncthreads();

for (i=0; i<index; i++) 
{ 
sum += sh_arr[i]; 
}
	average = sum / (index + 1.0f);
	printf("Thread id = %d\t Average = %f\n",index,average);

	if (array[index] > average) 
	{array[index] = average; }
	
	sh_arr[index] = 3.14;
}

int main(int argc, char **argv)
{
   LM<<<1, 128>>>(2.0f);

float h_arr[128];
float *d_arr;

hipMalloc((void **) &d_arr, sizeof(float) * 128);
hipMemcpy((void *)d_arr, (void *)h_arr, sizeof(float) * 128, hipMemcpyHostToDevice);
GM<<<1, 128>>>(d_arr);
hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128, hipMemcpyDeviceToHost);
SM<<<1, 128>>>(d_arr);
hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	return 0;
}
